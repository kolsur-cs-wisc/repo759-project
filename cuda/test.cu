#include "attention.cu"
#include <iostream>

#define L 3
#define D 2

int main() {
    // Host matrices
    float h_Q[L * D] = {1, 2, 3, 4, 5, 6};  // 2x3 matrix
    float h_K[L * D] = {7, 8, 9, 10, 11, 12};  // 2x3 matrix
    float h_V[L * D] = {1, 2, 3, 4, 5, 6};  // 2x3 matrix
    float h_output[L * D];
    
    // Device matrices
    float *d_Q, *d_K, *d_V, *d_output;
    hipMalloc(&d_Q, L * D * sizeof(float));
    hipMalloc(&d_K, L * D * sizeof(float));
    hipMalloc(&d_V, L * D * sizeof(float));
    hipMalloc(&d_output, L * D * sizeof(float));
    
    // Copy data to device
    hipMemcpy(d_Q, h_Q, L * D * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_K, h_K, L * D * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_V, h_V, L * D * sizeof(float), hipMemcpyHostToDevice);

    attention_forward(d_Q, d_K, d_V, d_output, L, D);

    hipMemcpy(h_output, d_output, L * D * sizeof(float), hipMemcpyDeviceToHost);
    
    std::cout << "Result matrix:" << std::endl;
    for (int i = 0; i < L; ++i) {
        for (int j = 0; j < D; ++j) {
            std::cout << h_output[i * D + j] << " ";
        }
        std::cout << std::endl;
    }
    
    // Free device memory
    hipFree(d_Q);
    hipFree(d_K);
    hipFree(d_V);
    hipFree(d_output);
    
    return 0;
    
    }
    